//
// Created by progamers on 7/18/25.
//
#include <thrust/sort.h>
#include <thrust/system/cuda/memory_resource.h>

#include "cuda_types/buffer.h"
#include "cuda_types/error.h"
#include "cuda_types/stream.h"
#include "graphics/vertex.h"
#include "sphere_generation/kernel_launcher.h"
#include "sphere_generation/tessellation_kernel.h"

namespace raw::sphere_generation {
/**
 * @brief Strict ordering comparator for edges (ascending by endpoints).
 *
 * Compares two edges first by their v0 index, and if those are equal, by their v1 index.
 * Designed to be usable on both host and device for sorting operations (e.g., with Thrust).
 *
 * @param a First edge to compare.
 * @param b Second edge to compare.
 * @return true if edge `a` should come before edge `b` (a < b) under the ordering.
 */
__device__ __host__ bool edge_comparator(const edge &a, const edge &b) {
	if (a.v0 > b.v0)
		return false;
	if (a.v0 < b.v0)
		return true;
	return a.v1 < b.v1;
}
/**
 * @brief Perform iterative triangular mesh tessellation (subdivision) on the device.
 *
 * Performs `steps` iterations of midpoint subdivision: for each iteration it generates edges
 * from the current triangle index buffer, creates unique midpoint vertices for edges,
 * rebuilds the triangle index buffer (each triangle becomes 4), and updates device-side
 * vertex/triangle counts. After the loop it computes per-vertex tangents/bitangents/UVs.
 *
 * The function operates entirely on device memory and CUDA streams: it launches device
 * kernels, performs device-to-device and device-to-host count transfers, and synchronizes
 * the provided CUDA stream at key points. If `steps` is odd, the final vertex/index buffers
 * are copied back into the original input buffers so the caller's base buffers contain the
 * final tessellated mesh.
 *
 * @param in_vertices Device pointer to the current input vertex buffer (may be swapped internally).
 * @param in_indices Device pointer to the current input index buffer (may be swapped internally).
 * @param all_edges Device array used to store generated edges for the current iteration.
 * @param out_vertices Device pointer to the output vertex buffer (used as a swap target).
 * @param out_indices Device pointer to the output index buffer (used as a swap target).
 * @param d_unique_edges Device array used to store unique edges identified for midpoint creation.
 * @param edge_to_vertex Device array mapping an edge key to its midpoint vertex index.
 * @param p_vertex_count Device pointer to a uint32_t counter holding the current vertex count;
 *        updated on-device as midpoints are created. (Device-side writable.)
 * @param p_triangle_count Device pointer to a uint32_t counter holding the current triangle count;
 *        the function multiplies this count by 4 each subdivision iteration. (Device-side writable.)
 * @param p_unique_edges_count Device pointer to a uint32_t used to return the number of unique edges
 *        discovered in each iteration; the host-side copy of this value is used to size subsequent sorts.
 * @param stream CUDA stream on which all operations and kernel launches are enqueued.
 * @param steps Number of subdivision iterations to perform.
 */
void launch_tessellation(raw::graphics::vertex *in_vertices, UI *in_indices, edge *all_edges,

						 raw::graphics::vertex *out_vertices, UI *out_indices, edge *d_unique_edges,
						 uint32_t *edge_to_vertex, uint32_t *p_vertex_count,
						 uint32_t *p_triangle_count, uint32_t *p_unique_edges_count,
						 hipStream_t &stream, uint32_t steps) {
	auto		   base_in_vertices	 = in_vertices;
	const auto	   base_in_indices	 = in_indices;
	uint32_t	   num_vertices_cpu	 = 12;
	constexpr auto threads_per_block = 1024;

	cuda_types::cuda_buffer<uint32_t, cuda_types::side::host> num_triangles_cpu(sizeof(uint32_t));
	cuda_types::cuda_buffer<uint32_t, cuda_types::side::host> num_unique_edges_cpu(
		sizeof(uint32_t));
	*num_triangles_cpu = predef::BASIC_AMOUNT_OF_TRIANGLES;
	auto blocks		   = (*num_triangles_cpu + threads_per_block - 1) / threads_per_block;
	int	 base		   = 12;
	CUDA_SAFE_CALL(
		hipMemcpyAsync(p_vertex_count, &base, sizeof(uint32_t), hipMemcpyHostToDevice, stream));
	// This shit right here, took me fckn 2 hours to understand LOL
	for (uint32_t i = 0; i < steps; ++i) {
		// we need to sync here for above operation to finish, get correct "num_vertices_cpu" and
		// then perform bottom operation with correct byte size
		if (i != 0) {
			CUDA_SAFE_CALL(
				hipMemcpyAsync(out_vertices, in_vertices,
								predef::MAXIMUM_AMOUNT_OF_VERTICES * sizeof(raw::graphics::vertex),
								hipMemcpyDeviceToDevice, stream));
		} else {
			CUDA_SAFE_CALL(hipMemcpyAsync(out_vertices, in_vertices,
										   12 * sizeof(raw::graphics::vertex),
										   hipMemcpyDeviceToDevice, stream));
		}

		CUDA_SAFE_CALL(hipMemsetAsync(p_unique_edges_count, 0, sizeof(uint32_t), stream));

		blocks = (*num_triangles_cpu + threads_per_block - 1) / threads_per_block;
		generate_edges<<<blocks, threads_per_block, 0, stream>>>(in_indices, all_edges,
																 *num_triangles_cpu);
		thrust::sort(thrust::cuda::par_nosync.on(stream), all_edges,
					 all_edges + *num_triangles_cpu * 3);

		blocks = (*num_triangles_cpu * 3 + threads_per_block - 1) / threads_per_block;
		create_unique_midpoint_vertices<<<blocks, threads_per_block, 0, stream>>>(
			all_edges, in_vertices, out_vertices, p_vertex_count, d_unique_edges, edge_to_vertex,
			p_unique_edges_count, *num_triangles_cpu * 3);

		CUDA_SAFE_CALL(hipMemcpyAsync(num_unique_edges_cpu.get(), p_unique_edges_count,
									   sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
		CUDA_SAFE_CALL(hipStreamSynchronize(stream));
		thrust::sort_by_key(thrust::cuda::par_nosync.on(stream), d_unique_edges,
							d_unique_edges + *num_unique_edges_cpu, edge_to_vertex);

		blocks = (*num_triangles_cpu + threads_per_block - 1) / threads_per_block;

		create_triangles<<<blocks, threads_per_block, 0, stream>>>(
			in_indices, out_indices, d_unique_edges, edge_to_vertex, p_unique_edges_count,
			*num_triangles_cpu);

		*num_triangles_cpu *= 4;
		std::swap(in_vertices, out_vertices);
		std::swap(in_indices, out_indices);
	}
	in_vertices = base_in_vertices;
	in_indices	= base_in_indices;
	hipMemcpyAsync(&num_vertices_cpu, p_vertex_count, sizeof(uint32_t), hipMemcpyDeviceToHost,
					stream);
	hipStreamSynchronize(stream);
	if (steps % 2 != 0) {
		hipMemcpyAsync(base_in_vertices, out_vertices, num_vertices_cpu * sizeof(graphics::vertex),
						hipMemcpyDeviceToDevice, stream);
		hipMemcpyAsync(base_in_indices, out_indices, *num_triangles_cpu * 3 * sizeof(UI),
						hipMemcpyDeviceToDevice, stream);
	}
	blocks = (num_vertices_cpu + threads_per_block - 1) / threads_per_block;

	calculate_tbn_and_uv<<<blocks, threads_per_block, 0, stream>>>(in_vertices, num_vertices_cpu);
	CUDA_SAFE_CALL(hipStreamSynchronize(stream));
}

} // namespace raw::sphere_generation