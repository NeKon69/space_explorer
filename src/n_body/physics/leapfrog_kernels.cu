#include "hip/hip_runtime.h"
//
// Created by progamers on 7/21/25.
//

#include <cstdio>
#include <glm/gtc/matrix_transform.hpp>

#include "n_body/cuda/physics/leapfrog_kernels.h"
#include "n_body/physics_component.h"
namespace raw::n_body::cuda::physics {
template<typename T>
__device__ void print_matrix_device(const glm::mat4 &m, const char *label) {
	printf("--- %s (Device) ---\n", label);
	for (int i = 0; i < 4; ++i) {
		printf("[ %10.4f %10.4f %10.4f %10.4f ]\n", m[0][i], m[1][i], m[2][i], m[3][i]);
	}
	printf("-------------------------\n");
}

template<typename T>
__device__ void compute_kick_rw(const space_object_data<T> *read_ptr,
								space_object_data<T> *write_ptr, uint16_t count, uint16_t current,
								T g, T epsilon, T dt) {
	auto						a_total		   = glm::vec<3, T>(0.0);
	const space_object_data<T> &current_object = read_ptr[current];
	// Using short here, since we don't plan on 5 billion planets to interact, maximum 10k
	for (uint16_t i = 0; i < count; ++i) {
		// Don't apply acceleration of itself
		if (current == i) {
			continue;
		}
		const space_object_data<T> &local_object = read_ptr[i];
		glm::vec<3, T>				dist		 = local_object.position - current_object.position;
		T							dist_sq	 = dist.x * dist.x + dist.y * dist.y + dist.z * dist.z;
		T							inv_dist = rsqrt(dist_sq + epsilon * epsilon);
		T							inv_dist_cubed = inv_dist * inv_dist * inv_dist;
		a_total += (g * local_object.mass * inv_dist_cubed) * dist;
	}

	write_ptr[current].velocity += a_total * dt;
	if (write_ptr != read_ptr) {
		write_ptr[current].position = read_ptr[current].position;
	}
}

template<typename T>
__device__ void compute_kick(const space_object_data<T> *object_old, space_object_data<T> *objects,
							 uint16_t count, uint16_t current, T g, T epsilon, T dt) {
	compute_kick_rw(object_old, objects, count, current, g, epsilon, dt);
}
template<typename T>
__device__ void compute_kick(space_object_data<T> *objects, uint16_t count, uint16_t current, T g,
							 T epsilon, T dt) {
	compute_kick_rw(objects, objects, count, current, g, epsilon, dt);
}

template<typename T>
__global__ void compute_k(graphics::instanced_data *data, const space_object_data<T> *object_old,
						  space_object_data<T> *objects, uint16_t count, T dt, T g, T epsilon) {
	const uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= count) {
		return;
	}

	// Kick
	compute_kick<T>(object_old, objects, count, x, g, epsilon, dt / 2);
}
template<typename T>
__global__ void compute_d(space_object_data<T> *objects, uint16_t count, T dt) {
	const uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= count) {
		return;
	}

	objects[x].position += dt * objects[x].velocity;
}

template<typename T>
__global__ void compute_k_final(graphics::instanced_data *data, space_object_data<T> *objects,
								uint16_t count, T dt, T g, T epsilon) {
	const uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= count) {
		return;
	}

	compute_kick<T>(objects, count, x, g, epsilon, dt / 2);

	data[x].model = glm::mat4(1.0f);

	data[x].model =
		glm::scale(glm::translate(glm::mat4(1.0f), static_cast<glm::vec3>(objects[x].position)),
				   glm::vec3(objects[x].radius));
}

template __global__ void compute_k<float>(graphics::instanced_data *data,
										  space_object_data<float> *objects, uint16_t count,
										  float dt, float g, float epsilon);

template __global__ void compute_k<double>(graphics::instanced_data	 *data,
										   space_object_data<double> *objects, uint16_t count,
										   double dt, double g, double epsilon);
template __global__ void compute_d<float>(space_object_data<float> *objects, uint16_t count,
										  float dt);
template __global__ void compute_d<double>(space_object_data<double> *objects, uint16_t count,
										   double dt);
template __global__ void compute_k_final<float>(graphics::instanced_data *data,
												space_object_data<float> *objects, uint16_t count,
												float dt, float g, float epsilon);
template __global__ void compute_k_final<double>(graphics::instanced_data  *data,
												 space_object_data<double> *objects, uint16_t count,
												 double dt, double g, double epsilon);

} // namespace raw::n_body::cuda::physics