#include "hip/hip_runtime.h"
//
// Created by progamers on 7/18/25.
//
#include "sphere_generation/tessellation_kernel.h"
#ifndef HIP_PI_F
#define HIP_PI_F 3.141592654f
#endif
__device__ void calc_tex_coords(glm::vec2* writing_ptr, glm::vec3& normalized_pos) {
	float u		 = atan2f(normalized_pos.z, normalized_pos.x) / (2.0f * HIP_PI_F) + 0.5f;
	float v		 = 0.5f - asinf(normalized_pos.y) / HIP_PI_F;
	*writing_ptr = glm::vec2 {u, v};
}

__global__ void subdivide(const glm::vec3* in_vertices, const unsigned int* in_indices,
                          glm::vec3* out_vertices,  glm::vec2* out_tex_coords,
                          unsigned int* out_indices, uint32_t* p_vertex_count,
                          uint32_t* p_triangle_count, size_t num_input_triangles) {
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= num_input_triangles) {
		return;
	}

	size_t i0 = in_indices[x * 3 + 0];
	size_t i1 = in_indices[x * 3 + 1];
	size_t i2 = in_indices[x * 3 + 2];

	glm::vec3 v0 = in_vertices[i0];
	glm::vec3 v1 = in_vertices[i1];
	glm::vec3 v2 = in_vertices[i2];

	glm::vec3 m01 = normalize(v0 + v1);
	glm::vec3 m12 = normalize(v1 + v2);
	glm::vec3 m20 = normalize(v2 + v0);

	uint32_t base_v_idx = atomicAdd(p_vertex_count, 3);
	uint32_t new_i01	= base_v_idx + 0;
	uint32_t new_i12	= base_v_idx + 1;
	uint32_t new_i20	= base_v_idx + 2;

	out_vertices[new_i01] = normalize(m01);
	calc_tex_coords(&out_tex_coords[new_i01], out_vertices[new_i01]);
	out_vertices[new_i12] = normalize(m12);
	calc_tex_coords(&out_tex_coords[new_i12], out_vertices[new_i12]);
	out_vertices[new_i20] = normalize(m20);
	calc_tex_coords(&out_tex_coords[new_i20], out_vertices[new_i20]);

	uint32_t	  base_t_idx  = atomicAdd(p_triangle_count, 4);
	unsigned int* out_tri_ptr = &out_indices[base_t_idx * 3];

	out_tri_ptr[0]	= i0;
	out_tri_ptr[1]	= new_i01;
	out_tri_ptr[2]	= new_i20;
	out_tri_ptr[3]	= i1;
	out_tri_ptr[4]	= new_i12;
	out_tri_ptr[5]	= new_i01;
	out_tri_ptr[6]	= i2;
	out_tri_ptr[7]	= new_i20;
	out_tri_ptr[8]	= new_i12;
	out_tri_ptr[9]	= new_i01;
	out_tri_ptr[10] = new_i12;
	out_tri_ptr[11] = new_i20;
}